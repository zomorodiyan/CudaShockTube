#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <assert.h>
#include <iostream>

#include "ShockTube.cuh"

//#include <algorithm> // in order to use std::max and std::min

#define fail "\033[1;31m"
#define reset "\033[0m"
#define cudaErrorCheck(call)                                \
{                                                           \
  hipError_t cucheck_err = (call);                         \
  if(cucheck_err != hipSuccess) {                          \
    const char *err_str = hipGetErrorString(cucheck_err);  \
    printf("\n%s (%d): %s%s%s\n", __FILE__, __LINE__, fail, err_str, reset);\
    assert(0);                                              \
  }                                                         \
}
// Wrap device CUDA calls with cucheck_err as in the following example.
// cudaErrorCheck(hipGetLastError());

// Allocate space for device copies of the variables
void ShockTube::allocDeviceMemory() {
	int size = nbrOfGrids * sizeof(double);
	cudaErrorCheck(hipMalloc((void **)&d_u1, size));
	cudaErrorCheck(hipMalloc((void **)&d_u2, size));
	cudaErrorCheck(hipMalloc((void **)&d_u3, size));
	cudaErrorCheck(hipMalloc((void **)&d_f1, size));
	cudaErrorCheck(hipMalloc((void **)&d_f2, size));
	cudaErrorCheck(hipMalloc((void **)&d_f3, size));
	cudaErrorCheck(hipMalloc((void **)&d_vol, size));
	cudaErrorCheck(hipMalloc((void **)&d_h, sizeof(double)));
	cudaErrorCheck(hipMalloc((void **)&d_length, sizeof(double)));
	cudaErrorCheck(hipMalloc((void **)&d_gama, sizeof(double)));
	cudaErrorCheck(hipMalloc((void **)&d_cfl, sizeof(double)));
	cudaErrorCheck(hipMalloc((void **)&d_nu, sizeof(double)));
	cudaErrorCheck(hipMalloc((void **)&d_tau, sizeof(double)));
	cudaErrorCheck(hipMalloc((void **)&d_cMax, sizeof(double)));
	cudaErrorCheck(hipMalloc((void **)&d_t, sizeof(double)));
		// only used in Lax-Wendroff step
	cudaErrorCheck(hipMalloc((void **)&d_u1Temp, size));
	cudaErrorCheck(hipMalloc((void **)&d_u2Temp, size));
	cudaErrorCheck(hipMalloc((void **)&d_u3Temp, size));
		// only used in Roe step
	cudaErrorCheck(hipMalloc((void **)&w1, size));
	cudaErrorCheck(hipMalloc((void **)&w2, size));
	cudaErrorCheck(hipMalloc((void **)&w3, size));
	cudaErrorCheck(hipMalloc((void **)&w4, size));
	cudaErrorCheck(hipMalloc((void **)&fc1, size));
	cudaErrorCheck(hipMalloc((void **)&fc2, size));
	cudaErrorCheck(hipMalloc((void **)&fc3, size));
	cudaErrorCheck(hipMalloc((void **)&fr1, size));
	cudaErrorCheck(hipMalloc((void **)&fr2, size));
	cudaErrorCheck(hipMalloc((void **)&fr3, size));
	cudaErrorCheck(hipMalloc((void **)&fl1, size));
	cudaErrorCheck(hipMalloc((void **)&fl2, size));
	cudaErrorCheck(hipMalloc((void **)&fl3, size));
	cudaErrorCheck(hipMalloc((void **)&fludif1, size));
	cudaErrorCheck(hipMalloc((void **)&fludif2, size));
	cudaErrorCheck(hipMalloc((void **)&fludif3, size));
	cudaErrorCheck(hipMalloc((void **)&eiglam1, size));
	cudaErrorCheck(hipMalloc((void **)&eiglam2, size));
	cudaErrorCheck(hipMalloc((void **)&eiglam3, size));
	cudaErrorCheck(hipMalloc((void **)&sgn1, size));
	cudaErrorCheck(hipMalloc((void **)&sgn2, size));
	cudaErrorCheck(hipMalloc((void **)&sgn3, size));
	cudaErrorCheck(hipMalloc((void **)&a1, size));
	cudaErrorCheck(hipMalloc((void **)&a2, size));
	cudaErrorCheck(hipMalloc((void **)&a3, size));
	cudaErrorCheck(hipMalloc((void **)&ac11, size));
	cudaErrorCheck(hipMalloc((void **)&ac12, size));
	cudaErrorCheck(hipMalloc((void **)&ac13, size));
	cudaErrorCheck(hipMalloc((void **)&ac21, size));
	cudaErrorCheck(hipMalloc((void **)&ac22, size));
	cudaErrorCheck(hipMalloc((void **)&ac23, size));
	cudaErrorCheck(hipMalloc((void **)&rsumr, size));
	cudaErrorCheck(hipMalloc((void **)&utilde, size));
	cudaErrorCheck(hipMalloc((void **)&htilde, size));
	cudaErrorCheck(hipMalloc((void **)&uvdif, size));
	cudaErrorCheck(hipMalloc((void **)&absvt, size));
	cudaErrorCheck(hipMalloc((void **)&ssc, size));
	cudaErrorCheck(hipMalloc((void **)&vsc, size));
	cudaErrorCheck(hipMalloc((void **)&isb1, nbrOfGrids * sizeof(int)));
	cudaErrorCheck(hipMalloc((void **)&isb2, nbrOfGrids * sizeof(int)));
	cudaErrorCheck(hipMalloc((void **)&isb3, nbrOfGrids * sizeof(int)));
}

// Free allocated space for device copies of the variables
void ShockTube::freeDeviceMemory() {
	cudaErrorCheck(hipFree(d_u1));
	cudaErrorCheck(hipFree(d_u2));
	cudaErrorCheck(hipFree(d_u3));
	cudaErrorCheck(hipFree(d_f1));
	cudaErrorCheck(hipFree(d_f2));
	cudaErrorCheck(hipFree(d_f3));
	cudaErrorCheck(hipFree(d_vol));
	cudaErrorCheck(hipFree(d_h));
	cudaErrorCheck(hipFree(d_length));
	cudaErrorCheck(hipFree(d_gama));
	cudaErrorCheck(hipFree(d_cfl));
	cudaErrorCheck(hipFree(d_nu));
	cudaErrorCheck(hipFree(d_tau));
	cudaErrorCheck(hipFree(d_cMax));
		// only used in Lax-Wendroff step
	cudaErrorCheck(hipFree(d_u1Temp));
	cudaErrorCheck(hipFree(d_u2Temp));
	cudaErrorCheck(hipFree(d_u3Temp));
		// only used in Roe step 
	cudaErrorCheck(hipFree(w1));
	cudaErrorCheck(hipFree(w2));
	cudaErrorCheck(hipFree(w3));
	cudaErrorCheck(hipFree(w4));
	cudaErrorCheck(hipFree(fc1)); 
	cudaErrorCheck(hipFree(fc2));
	cudaErrorCheck(hipFree(fc3));
	cudaErrorCheck(hipFree(fr1)); 
	cudaErrorCheck(hipFree(fr2)); 
	cudaErrorCheck(hipFree(fr3));
	cudaErrorCheck(hipFree(fl1));
	cudaErrorCheck(hipFree(fl2));
	cudaErrorCheck(hipFree(fl3));
	cudaErrorCheck(hipFree(fludif1)); 
	cudaErrorCheck(hipFree(fludif2)); 
	cudaErrorCheck(hipFree(fludif3));
	cudaErrorCheck(hipFree(eiglam1));
	cudaErrorCheck(hipFree(eiglam2)); 
	cudaErrorCheck(hipFree(eiglam3));
	cudaErrorCheck(hipFree(sgn1)); 
	cudaErrorCheck(hipFree(sgn2)); 
	cudaErrorCheck(hipFree(sgn3));
	cudaErrorCheck(hipFree(isb1)); 
	cudaErrorCheck(hipFree(isb2)); 
	cudaErrorCheck(hipFree(isb3));
	cudaErrorCheck(hipFree(a1)); 
	cudaErrorCheck(hipFree(a2)); 
	cudaErrorCheck(hipFree(a3));
	cudaErrorCheck(hipFree(ac11)); 
	cudaErrorCheck(hipFree(ac12)); 
	cudaErrorCheck(hipFree(ac13));
	cudaErrorCheck(hipFree(ac21)); 
	cudaErrorCheck(hipFree(ac22)); 
	cudaErrorCheck(hipFree(ac23));
	cudaErrorCheck(hipFree(rsumr));
	cudaErrorCheck(hipFree(utilde));
	cudaErrorCheck(hipFree(htilde));
	cudaErrorCheck(hipFree(uvdif));
	cudaErrorCheck(hipFree(absvt));
	cudaErrorCheck(hipFree(ssc));
	cudaErrorCheck(hipFree(vsc));
}

// calculate and update value of d_cMax
__device__ void updateCMax(const int nbrOfGrids, const double *d_u1, 
	const double *d_u2, const double *d_u3, const double *d_gama, double *d_cMax) 
{ 
	*d_cMax = 0; int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	double ro, p, u;
	__shared__ double c;
	for (int i = index; i < nbrOfGrids; i += stride){
		if (d_u1[i] == 0)
			continue;
		ro = d_u1[i];
		u = d_u2[i] / ro;
		p = (d_u3[i] - ro * u * u / 2) * (*d_gama - 1);
		c = sqrt(*d_gama * abs(p) / ro);
		if (*d_cMax < c + abs(u))
			*d_cMax = c + abs(u);
	}
}

// Assign Sod's shock tube problem initial conditions to device memory
__global__ void initDeviceMemory(const int nbrOfGrids, double *d_u1,
	double *d_u2, double *d_u3, double *d_vol, double *d_h,
	double *d_length, double *d_gama, double *d_cfl, double *d_nu,
	double *d_tau, double *d_cMax, double *d_t) {
	*d_t = 0;								// time
	*d_length = 1;							// length of shock tube
	*d_gama = 1.4;							// ratio of specific heats
	*d_cfl = 0.9;							// Courant-Friedrichs-Lewy number
	*d_nu = 0.0;							// artificial viscosity coefficient
	*d_h = *d_length / (nbrOfGrids - 1);	// space grid size
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x;
	for(int i = index; i < nbrOfGrids; i+= stride){
		double e, ro, p, u = 0;
		if (i < nbrOfGrids){
			if (i >= int(nbrOfGrids / 2)) { ro = 0.125, p = 0.1; }
			else { ro = 1, p = 1; }
			e = p / (*d_gama - 1) + ro * u * u / 2;
			d_u1[i] = ro;
			d_u2[i] = ro * u;
			d_u3[i] = e;
			d_u3[i] = e;
			d_vol[i] = 1;
		}
	}
	updateCMax(nbrOfGrids, d_u1, d_u2, d_u3, d_gama, d_cMax); 
	*d_tau = (*d_cfl) * (*d_h) / (*d_cMax);    // initial time grid size, It will be modified to tMax if this > tMax
}

// copy device data members to host data members
void ShockTube::copyDeviceToHost(const int nbrOfGrids) {
	int size = nbrOfGrids * sizeof(double);
	cudaErrorCheck(hipMemcpy(u1, d_u1, size, hipMemcpyDeviceToHost));
	cudaErrorCheck(hipMemcpy(u2, d_u2, size, hipMemcpyDeviceToHost));
	cudaErrorCheck(hipMemcpy(u3, d_u3, size, hipMemcpyDeviceToHost));
}

// copy host data members to device data members
void ShockTube::copyHostToDevice(const int nbrOfGrids) {
	int size = nbrOfGrids * sizeof(double);
	cudaErrorCheck(hipMemcpy(d_u1, u1, size, hipMemcpyHostToDevice));
	cudaErrorCheck(hipMemcpy(d_u2, u2, size, hipMemcpyHostToDevice));
	cudaErrorCheck(hipMemcpy(d_u3, u3, size, hipMemcpyHostToDevice));
}

__global__ void updateTau(const int nbrOfGrids, const double *d_u1,
	const double *d_u2, const double *d_u3, const double *d_gama,
	double *d_cMax, const double *d_h, const double *d_cfl, double *d_tau) {
	updateCMax(nbrOfGrids, d_u1, d_u2, d_u3, d_gama, d_cMax);
	*d_tau = *d_cfl * *d_h / *d_cMax;
}

 __global__ void boundaryCondition(const int nbrOfGrids,
	double *d_u1, double *d_u2, double *d_u3) {
	d_u1[0] = d_u1[1];
	d_u2[0] = -d_u2[1];
	d_u3[0] = d_u3[1];
	d_u1[nbrOfGrids - 1] = d_u1[nbrOfGrids - 2];
	d_u2[nbrOfGrids - 1] = -d_u2[nbrOfGrids - 2];
	d_u3[nbrOfGrids - 1] = d_u3[nbrOfGrids - 2];
}

// used in laxWendroffStep 
__device__ void d_boundaryCondition(const int nbrOfGrids,
	double *d_u1, double *d_u2, double *d_u3) {
	d_u1[0] = d_u1[1];
	d_u2[0] = -d_u2[1];
	d_u3[0] = d_u3[1];
	d_u1[nbrOfGrids - 1] = d_u1[nbrOfGrids - 2];
	d_u2[nbrOfGrids - 1] = -d_u2[nbrOfGrids - 2];
	d_u3[nbrOfGrids - 1] = d_u3[nbrOfGrids - 2];
}

// used in laxWendroffStep
__device__ void updateFlux(const int nbrOfGrids, const double *d_u1, const double *d_u2,
	const double *d_u3, double *d_f1, double *d_f2, double *d_f3, const double *d_gama) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x;
	double rho, m, e, p;
	for (int i = index; i < nbrOfGrids; i += stride) {
		rho = d_u1[i];
		m = d_u2[i];
		e = d_u3[i];
		p = (*d_gama - 1) * (e - m * m / rho / 2);
		d_f1[i] = m;
		d_f2[i] = m * m / rho + p;
		d_f3[i] = m / rho * (e + p);
	}
}

// used in laxWendroffStep
__device__ void halfStep(const int nbrOfGrids, const double *d_u1, const double *d_u2,
	const double *d_u3, double *d_u1Temp, double *d_u2Temp, double *d_u3Temp,
	const double *d_f1, const double *d_f2, const double *d_f3, const double *d_tau, const double *d_h) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x;
	for (int i = index; i < nbrOfGrids; i += stride) {
		if ((i > 0) && (i < nbrOfGrids - 1)) {
			d_u1Temp[i] = (d_u1[i + 1] + d_u1[i]) / 2 - *d_tau / 2 / *d_h * (d_f1[i + 1] - d_f1[i]);
			d_u2Temp[i] = (d_u2[i + 1] + d_u2[i]) / 2 - *d_tau / 2 / *d_h * (d_f2[i + 1] - d_f2[i]);
			d_u3Temp[i] = (d_u3[i + 1] + d_u3[i]) / 2 - *d_tau / 2 / *d_h * (d_f3[i + 1] - d_f3[i]);
		}
	}
}

// used in laxWendroffStep
__device__ void step(const int nbrOfGrids, const double *d_u1, const double *d_u2,
	const double *d_u3, double *d_u1Temp, double *d_u2Temp, double *d_u3Temp,
	const double *d_f1, const double *d_f2, const double *d_f3, const double *d_tau, const double *d_h) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x;
	for (int i = index; i < nbrOfGrids; i += stride) {
		if ((i > 0) && (i < nbrOfGrids - 1)) {
			d_u1Temp[i] = d_u1[i] - *d_tau / *d_h * (d_f1[i] - d_f1[i - 1]);
			d_u2Temp[i] = d_u2[i] - *d_tau / *d_h * (d_f2[i] - d_f2[i - 1]);
			d_u3Temp[i] = d_u3[i] - *d_tau / *d_h * (d_f3[i] - d_f3[i - 1]);
		}
	}
}

// used in laxWendroffStep
__device__ void updateU(const int nbrOfGrids, double *d_u1, double *d_u2,
	double *d_u3, const double *d_u1Temp, const double *d_u2Temp, const double *d_u3Temp) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x;
	for (int i = index; i < nbrOfGrids; i += stride) {
		if ((i > 0) && (i < nbrOfGrids - 1)) {
			d_u1[i] = d_u1Temp[i];
			d_u2[i] = d_u2Temp[i];
			d_u3[i] = d_u3Temp[i];
		}
	}
}

__global__	void laxWendroffStep(const int nbrOfGrids, double *d_u1, double *d_u2,
	double *d_u3, double *d_u1Temp, double *d_u2Temp, double *d_u3Temp,
	double *d_f1, double *d_f2, double *d_f3, const double *d_tau, const double *d_h, const double *d_gama) {
	updateFlux(nbrOfGrids, d_u1, d_u2, d_u3, d_f1, d_f2, d_f3, d_gama);
	halfStep(nbrOfGrids, d_u1, d_u2, d_u3, d_u1Temp, d_u2Temp, d_u3Temp, d_f1, d_f2, d_f3, d_tau, d_h);
	d_boundaryCondition(nbrOfGrids, d_u1Temp, d_u2Temp, d_u3Temp);
	updateFlux(nbrOfGrids, d_u1Temp, d_u2Temp, d_u3Temp, d_f1, d_f2, d_f3, d_gama);
	step(nbrOfGrids, d_u1, d_u2, d_u3, d_u1Temp, d_u2Temp, d_u3Temp, d_f1, d_f2, d_f3, d_tau, d_h);
	updateU(nbrOfGrids, d_u1, d_u2, d_u3, d_u1Temp, d_u2Temp, d_u3Temp);
	d_boundaryCondition(nbrOfGrids, d_u1, d_u2, d_u3);
}

// used in RoeStep
	#define tiny 1e-30
	#define sbpar1 2.0
	#define sbpar2 2.0

__global__	void RoeStep(const int nbrOfGrids, double *d_u1, double *d_u2,
	double *d_u3, const double *d_vol, double *d_f1, double *d_f2, double *d_f3, 
	const double *d_tau, const double *d_h, const double *d_gama,
	double *w1,double *w2,double *w3,double *w4, double *fc1,double *fc2,double *fc3,
	double *fr1,double *fr2,double *fr3, double *fl1,double *fl2,double *fl3,
	double *fludif1,double *fludif2,double *fludif3,
	double *rsumr, double *utilde, double *htilde, double *uvdif, double *absvt, double *ssc, double *vsc,
	double *eiglam1,double *eiglam2,double *eiglam3, double *sgn1,double *sgn2,double *sgn3,
	int *isb1,int *isb2,int *isb3, double *a1,double *a2,double *a3,
	double *ac11,double *ac12,double *ac13, double *ac21,double *ac22,double *ac23) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x;
	for (int i = index; i < nbrOfGrids; i += stride) {

		// find parameter vector w
		{
			w1[i] = sqrt(d_vol[i] * d_u1[i]);
			w2[i] = w1[i] * d_u2[i] / d_u1[i];
			w4[i] = (*d_gama - 1) * (d_u3[i] - 0.5 * d_u2[i] * d_u2[i] / d_u1[i]);
			w3[i] = w1[i] * (d_u3[i] + w4[i]) / d_u1[i];
		}

		// calculate the fluxes at the cell center
		{
			fc1[i] = w1[i] * w2[i];
			fc2[i] = w2[i] * w2[i] + d_vol[i] * w4[i];
			fc3[i] = w2[i] * w3[i];
		}

		__syncthreads(); // because of the [i - 1] index below
		// calculate the fluxes at the cell walls 
		if (i > 0) {
			fl1[i] = fc1[i - 1]; fr1[i] = fc1[i];
			fl2[i] = fc2[i - 1]; fr2[i] = fc2[i];
			fl3[i] = fc3[i - 1]; fr3[i] = fc3[i];
		}

		// calculate the flux differences at the cell walls
		if (i > 0) {
			fludif1[i] = fr1[i] - fl1[i];
			fludif2[i] = fr2[i] - fl2[i];
			fludif3[i] = fr3[i] - fl3[i];
		}

		__syncthreads(); // because of the [i - 1] index below
		// calculate the tilded state variables = mean values at the interfaces
		if (i > 0) {
			rsumr[i] = 1 / (w1[i - 1] + w1[i]);

			utilde[i] = (w2[i - 1] + w2[i]) * rsumr[i];
			htilde[i] = (w3[i - 1] + w3[i]) * rsumr[i];

			absvt[i] = 0.5 * utilde[i] * utilde[i];
			uvdif[i] = utilde[i] * fludif2[i];

			ssc[i] = (*d_gama - 1) * (htilde[i] - absvt[i]);
			if (ssc[i] > 0.0)
				vsc[i] = sqrt(ssc[i]);
			else {
				vsc[i] = sqrt(abs(ssc[i]));
			}
		}

		// calculate the eigenvalues and projection coefficients for each eigenvector
		if (i > 0) {
			eiglam1[i] = utilde[i] - vsc[i];
			eiglam2[i] = utilde[i];
			eiglam3[i] = utilde[i] + vsc[i];
			sgn1[i] = eiglam1[i] < 0.0 ? -1 : 1;
			sgn2[i] = eiglam2[i] < 0.0 ? -1 : 1;
			sgn3[i] = eiglam3[i] < 0.0 ? -1 : 1;
			a1[i] = 0.5 * ((*d_gama - 1) * (absvt[i] * fludif1[i] + fludif3[i]
				- uvdif[i]) - vsc[i] * (fludif2[i] - utilde[i]
					* fludif1[i])) / ssc[i];
			a2[i] = (*d_gama - 1) * ((htilde[i] - 2 * absvt[i]) * fludif1[i]
				+ uvdif[i] - fludif3[i]) / ssc[i];
			a3[i] = 0.5 * ((*d_gama - 1) * (absvt[i] * fludif1[i] + fludif3[i]
				- uvdif[i]) + vsc[i] * (fludif2[i] - utilde[i]
					* fludif1[i])) / ssc[i];
		}

		// divide the projection coefficients by the wave speeds to evade expansion correction
		if (i > 0) {
			a1[i] /= eiglam1[i] + tiny;
			a2[i] /= eiglam2[i] + tiny;
			a3[i] /= eiglam3[i] + tiny;
		}

		// calculate the first order projection coefficients ac1
		if (i > 0) {
			ac11[i] = -sgn1[i] * a1[i] * eiglam1[i];
			ac12[i] = -sgn2[i] * a2[i] * eiglam2[i];
			ac13[i] = -sgn3[i] * a3[i] * eiglam3[i];
		}

		// apply the 'superbee' flux correction to made 2nd order projection coefficients ac2
		{
			ac21[1] = ac11[1];
			ac21[nbrOfGrids - 1] = ac11[nbrOfGrids - 1];
			ac22[1] = ac12[1];
			ac22[nbrOfGrids - 1] = ac12[nbrOfGrids - 1];
			ac23[1] = ac13[1];
			ac23[nbrOfGrids - 1] = ac13[nbrOfGrids - 1];


			double dtdx = *d_tau / *d_h;
			if ((i > 1) && (i < nbrOfGrids - 1)) {
				isb1[i] = i - int(sgn1[i]);
				ac21[i] = ac11[i] + eiglam1[i] *
					((fmax(0.0, fmin(sbpar1 * a1[isb1[i]], fmax(a1[i], fmin(a1[isb1[i]], sbpar2 * a1[i])))) +
						fmin(0.0, fmax(sbpar1 * a1[isb1[i]], fmin(a1[i], fmax(a1[isb1[i]], sbpar2 * a1[i]))))) *
						(sgn1[i] - dtdx * eiglam1[i]));
				isb2[i] = i - int(sgn2[i]);
				ac22[i] = ac12[i] + eiglam2[i] *
					((fmax(0.0, fmin(sbpar1 * a2[isb2[i]], fmax(a2[i], fmin(a2[isb2[i]], sbpar2 * a2[i])))) +
						fmin(0.0, fmax(sbpar1 * a2[isb2[i]], fmin(a2[i], fmax(a2[isb2[i]], sbpar2 * a2[i]))))) *
						(sgn2[i] - dtdx * eiglam2[i]));
				isb3[i] = i - int(sgn3[i]);
				ac23[i] = ac13[i] + eiglam3[i] *
					((fmax(0.0, fmin(sbpar1 * a3[isb3[i]], fmax(a3[i], fmin(a3[isb3[i]], sbpar2 * a3[i])))) +
						fmin(0.0, fmax(sbpar1 * a3[isb3[i]], fmin(a3[i], fmax(a3[isb3[i]], sbpar2 * a3[i]))))) *
						(sgn3[i] - dtdx * eiglam3[i]));
			}
		}

		// calculate the final fluxes
		if (i > 0) {
			d_f1[i] = 0.5 * (fl1[i] + fr1[i] + ac21[i] + ac22[i] + ac23[i]);
			d_f2[i] = 0.5 * (fl2[i] + fr2[i] + eiglam1[i] * ac21[i]
				+ eiglam2[i] * ac22[i] + eiglam3[i] * ac23[i]);
			d_f3[i] = 0.5 * (fl3[i] + fr3[i] + (htilde[i] - utilde[i] * vsc[i]) * ac21[i]
				+ absvt[i] * ac22[i] + (htilde[i] + utilde[i] * vsc[i]) * ac23[i]);
		}

		__syncthreads(); // because of the [i + 1] index below
		// update U
		if (i > 0 && i < nbrOfGrids - 1) {
			d_u1[i] -= *d_tau / *d_h * (d_f1[i + 1] - d_f1[i]);
			d_u2[i] -= *d_tau / *d_h * (d_f2[i + 1] - d_f2[i]);
			d_u3[i] -= *d_tau / *d_h * (d_f3[i + 1] - d_f3[i]);
		}

		d_boundaryCondition(nbrOfGrids, d_u1, d_u2, d_u3);
	}
}
