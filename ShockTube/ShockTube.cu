#include "hip/hip_runtime.h"
#include ""
#include <assert.h>
#include <iostream>
#include "ShockTube.cuh"


#define cucheck_dev(call)                                   \
{                                                           \
  hipError_t cucheck_err = (call);                         \
  if(cucheck_err != hipSuccess) {                          \
    const char *err_str = hipGetErrorString(cucheck_err);  \
    printf("%s (%d): %s\n", __FILE__, __LINE__, err_str);   \
    assert(0);                                              \
  }                                                         \
}
// Wrap device CUDA calls with cucheck_err as in the following example.
// cucheck_dev(hipGetLastError());


// Allocate space for device copies of the variables
void ShockTube::allocDeviceMemory() {
	int size = nbrOfGrids * sizeof(double);
	cucheck_dev(hipMalloc((void **)&d_u1, size));
	cucheck_dev(hipMalloc((void **)&d_u2, size));
	cucheck_dev(hipMalloc((void **)&d_u3, size));
	cucheck_dev(hipMalloc((void **)&d_f1, size));
	cucheck_dev(hipMalloc((void **)&d_f2, size));
	cucheck_dev(hipMalloc((void **)&d_f3, size));
	cucheck_dev(hipMalloc((void **)&d_vol, size));
}
