#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <assert.h>
#include <iostream>
#include "ShockTube.cuh"


#define fail "\033[1;31m"
#define reset "\033[0m"
#define cudaErrorCheck(call)                                \
{                                                           \
  hipError_t cucheck_err = (call);                         \
  if(cucheck_err != hipSuccess) {                          \
    const char *err_str = hipGetErrorString(cucheck_err);  \
    printf("\n%s (%d): %s%s%s\n", __FILE__, __LINE__, fail, err_str, reset);\
    assert(0);                                              \
  }                                                         \
}
// Wrap device CUDA calls with cucheck_err as in the following example.
// cudaErrorCheck(hipGetLastError());


// Allocate space for device copies of the variables
void ShockTube::allocDeviceMemory() {
	int size = nbrOfGrids * sizeof(double);
	cudaErrorCheck(hipMalloc((void **)&d_u1, size));
	cudaErrorCheck(hipMalloc((void **)&d_u2, size));
	cudaErrorCheck(hipMalloc((void **)&d_u3, size));
	cudaErrorCheck(hipMalloc((void **)&d_f1, size));
	cudaErrorCheck(hipMalloc((void **)&d_f2, size));
	cudaErrorCheck(hipMalloc((void **)&d_f3, size));
	cudaErrorCheck(hipMalloc((void **)&d_vol, size));
	cudaErrorCheck(hipMalloc((void **)&d_h, sizeof(double)));
	cudaErrorCheck(hipMalloc((void **)&d_length, sizeof(double)));
	cudaErrorCheck(hipMalloc((void **)&d_gama, sizeof(double)));
	cudaErrorCheck(hipMalloc((void **)&d_cfl, sizeof(double)));
	cudaErrorCheck(hipMalloc((void **)&d_nu, sizeof(double)));
	cudaErrorCheck(hipMalloc((void **)&d_tau, sizeof(double)));
	cudaErrorCheck(hipMalloc((void **)&d_cMax, sizeof(double)));
	cudaErrorCheck(hipMalloc((void **)&d_t, sizeof(double)));
}

// Free allocated space for device copies of the variables
void ShockTube::freeDeviceMemory() {
	cudaErrorCheck(hipFree(d_u1));
	cudaErrorCheck(hipFree(d_u2));
	cudaErrorCheck(hipFree(d_u3));
	cudaErrorCheck(hipFree(d_f1));
	cudaErrorCheck(hipFree(d_f2));
	cudaErrorCheck(hipFree(d_f3));
	cudaErrorCheck(hipFree(d_vol));
	cudaErrorCheck(hipFree(d_h));
	cudaErrorCheck(hipFree(d_length));
	cudaErrorCheck(hipFree(d_gama));
	cudaErrorCheck(hipFree(d_cfl));
	cudaErrorCheck(hipFree(d_nu));
	cudaErrorCheck(hipFree(d_tau));
	cudaErrorCheck(hipFree(d_cMax));
}

// calculate and update value of d_cMax
__device__ void updateCMax(const int nbrOfGrids, const double *d_u1,
	const double *d_u2, const double *d_u3, const double *d_gama,
	double *d_cMax) {
	*d_cMax = 0;
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < nbrOfGrids; i += stride){
		double ro, p, u = 0;
		if (d_u1[i] == 0)
			continue;
		ro = d_u1[i];
		u = d_u2[i] / ro;
		p = (d_u3[i] - ro * u * u / 2) * (*d_gama - 1);
		double c = sqrt(*d_gama * abs(p) / ro);
		if (*d_cMax < c + abs(u))
			*d_cMax = c + abs(u);
	}
}

// Assign Sod's shock tube problem initial conditions to device memory
__global__ void initDeviceMemory(const int nbrOfGrids, double *d_u1,
	double *d_u2, double *d_u3, double *d_vol, double *d_h,
	double *d_length, double *d_gama, double *d_cfl, double *d_nu,
	double *d_tau, double *d_cMax, double *d_t) {
	*d_t = 0;							// time
	*d_length = 1.0;					// length of shock tube
	*d_gama = 1.4;						// ratio of specific heats
	*d_cfl = 0.9;						// Courant-Friedrichs-Lewy number
	*d_nu = 0.0;							// artificial viscosity coefficient
	*d_h = *d_length / (nbrOfGrids - 1);  // space grid size
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x;
	for(int i = index; i < nbrOfGrids; i+= stride){
		double e, ro, p, u = 0;
		if (index < nbrOfGrids){
			if (index >= int(nbrOfGrids / 2)) { ro = 0.125, p = 0.1; }
			else { ro = 1, p = 1; }
			e = p / (*d_gama - 1) + ro * u * u / 2;
			d_u1[i] = ro;
			d_u2[i] = ro * u;
			d_u3[i] = e;
			d_vol[i] = 1;
			updateCMax(nbrOfGrids, d_u1, d_u2, d_u3, d_gama, d_cMax); 
			*d_tau = *d_cfl * *d_h / *d_cMax;    // time grid size
		}
	}
}

// copy device data members to host data members
void ShockTube::copyDeviceToHost(const int nbrOfGrids) {
	int size = nbrOfGrids * sizeof(double);
	cudaErrorCheck(hipMemcpy(u1, d_u1, size, hipMemcpyDeviceToHost));
	cudaErrorCheck(hipMemcpy(u2, d_u2, size, hipMemcpyDeviceToHost));
	cudaErrorCheck(hipMemcpy(u3, d_u3, size, hipMemcpyDeviceToHost));
	/*/
	cudaErrorCheck(hipMemcpy(f1, d_f1, size, hipMemcpyDeviceToHost));
	cudaErrorCheck(hipMemcpy(f2, d_f2, size, hipMemcpyDeviceToHost));
	cudaErrorCheck(hipMemcpy(f3, d_f3, size, hipMemcpyDeviceToHost));
	/**/
	cudaErrorCheck(hipMemcpy(vol, d_vol, size, hipMemcpyDeviceToHost));
	cudaErrorCheck(hipMemcpy(&h, d_h, sizeof(double), hipMemcpyDeviceToHost));
	cudaErrorCheck(hipMemcpy(&length, d_length, sizeof(double), hipMemcpyDeviceToHost));
	cudaErrorCheck(hipMemcpy(&gama, d_gama, sizeof(double), hipMemcpyDeviceToHost));
	cudaErrorCheck(hipMemcpy(&cfl, d_cfl, sizeof(double), hipMemcpyDeviceToHost));
	cudaErrorCheck(hipMemcpy(&nu, d_nu, sizeof(double), hipMemcpyDeviceToHost));
	cudaErrorCheck(hipMemcpy(&tau, d_tau, sizeof(double), hipMemcpyDeviceToHost));
}

// copy host data members to device data members
void ShockTube::copyHostToDevice(const int nbrOfGrids) {
	int size = nbrOfGrids * sizeof(double);
	cudaErrorCheck(hipMemcpy(d_u1, u1, size, hipMemcpyHostToDevice));
	cudaErrorCheck(hipMemcpy(d_u2, u2, size, hipMemcpyHostToDevice));
	cudaErrorCheck(hipMemcpy(d_u3, u3, size, hipMemcpyHostToDevice));
	/*/
	cudaErrorCheck(hipMemcpy(d_f1, f1, size, hipMemcpyHostToDevice));
	cudaErrorCheck(hipMemcpy(d_f2, f2, size, hipMemcpyHostToDevice));
	cudaErrorCheck(hipMemcpy(d_f3, f3, size, hipMemcpyHostToDevice));
	/**/
	cudaErrorCheck(hipMemcpy(d_vol, vol, size, hipMemcpyHostToDevice));
	cudaErrorCheck(hipMemcpy(d_h, &h, sizeof(double), hipMemcpyHostToDevice));
	cudaErrorCheck(hipMemcpy(d_length, &length, sizeof(double), hipMemcpyHostToDevice));
	cudaErrorCheck(hipMemcpy(d_gama, &gama, sizeof(double), hipMemcpyHostToDevice));
	cudaErrorCheck(hipMemcpy(d_cfl, &cfl, sizeof(double), hipMemcpyHostToDevice));
	cudaErrorCheck(hipMemcpy(d_nu, &nu, sizeof(double), hipMemcpyHostToDevice));
	cudaErrorCheck(hipMemcpy(d_tau, &tau, sizeof(double), hipMemcpyHostToDevice));
}


__global__ void DeviceBoundaryCondition(const int nbrOfGrids,
	double *d_u1, double *d_u2, double *d_u3) {
	d_u1[0] = d_u1[1];
	d_u2[0] = -d_u2[1];
	d_u3[0] = d_u3[1];
	d_u1[nbrOfGrids - 1] = d_u1[nbrOfGrids - 2];
	d_u2[nbrOfGrids - 1] = -d_u2[nbrOfGrids - 2];
	d_u3[nbrOfGrids - 1] = d_u3[nbrOfGrids - 2];
}


// calculate and update value of d_tau
__device__ void updateCMax(const int nbrOfGrids, const double *d_u1,
	const double *d_u2, const double *d_u3, const double *d_gama,
	double *d_cMax, double *d_tau, double *d_cfl) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < nbrOfGrids; i += stride){
		;
	}
}


__device__ void updateTau(const int nbrOfGrids, const double *d_u1,
	const double *d_u2, const double *d_u3, const double *d_gama,
	double *d_cMax, const double *d_h, const double *d_cfl, double *d_tau) {
	updateCMax(nbrOfGrids, d_u1, d_u2, d_u3, d_gama, d_cMax);
	*d_tau = *d_cfl * *d_h / *d_cMax;
}
