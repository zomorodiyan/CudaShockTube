#include "hip/hip_runtime.h"

#include <iostream>
#include <fstream>
#include <string>
#include "ShockTube.cuh"
#include <iomanip>      // std::setprecision

#define coutPericision 30
#define eps 1e-14
#define fail " \033[1;31m"
#define pass " \033[1;32m"
#define yellow " \033[1;33m"
#define blue " \033[1;34m"
#define reset "\033[0m"
#define cudaErrorCheck(call)                                \
{                                                           \
  hipError_t cucheck_err = (call);                         \
  if(cucheck_err != hipSuccess) {                          \
    const char *err_str = hipGetErrorString(cucheck_err);  \
    printf("\n%s (%d): %s%s%s\n", __FILE__, __LINE__, fail, err_str, reset);\
  }                                                         \
}
// Wrap device CUDA calls with cucheck_err as in the following example:
// cudaErrorCheck(hipGetLastError());


void ShockTube::DeviceTest01() {
	const std::string test = "Memory Allocation And Initialization";
	std::cout << yellow << __func__ << reset;
	nbrOfGrids = 10;
	allocDeviceMemory();
	initDeviceMemory<<<1,16>>>(nbrOfGrids, d_u1, d_u2, d_u3, d_vol, d_h, d_length,
		d_gama, d_cfl, d_nu, d_tau, d_cMax, d_t);
	cudaErrorCheck(hipDeviceSynchronize());
	allocHostMemory();
	copyDeviceToHost(nbrOfGrids);
	freeDeviceMemory();
	updateAverages();
	freeHostMemory();
	//std::cout << std::endl << Averages[0] << "  " << Averages[1] << "  " << Averages[2] << "  " << Averages[3] << std::endl; /**/
	if ((abs(roAverage - 0.5625) < eps)
		&& (abs(uAverage - 0) < eps)
		&& (abs(eAverage - 1.375) < eps)
		&& (abs(pAverage - 0.55) < eps))
	std::cout << pass << test << reset << std::endl;
	else
	std::cout << fail << test << reset << std::endl;
}

void ShockTube::DeviceTest02() {
	const std::string test = "Boundary Condition";
	std::cout << yellow << __func__ << reset;
	nbrOfGrids = 10;
	allocHostMemory();
	initHostMemory();
	u1[1] = u1[8] = 1; 	u2[1] = u2[8] = -1;	u3[1] = u3[8] = 1; 
	allocDeviceMemory();
	copyHostToDevice(nbrOfGrids);
	boundaryCondition<<<1,16>>>(nbrOfGrids, d_u1, d_u2, d_u3);
	copyDeviceToHost(nbrOfGrids);
	freeDeviceMemory();
	if((1 == u1[0]) && (1 == u1[9]) && (1 == u2[0])
		&& (1 == u2[9]) && (1 == u3[0]) && (1 == u3[9]))
		std::cout << pass << test << reset << std::endl;
	else
		std::cout << fail << test << reset << std::endl;
	freeHostMemory();
}

void ShockTube::DeviceTest03() {
	const std::string test = "LaxWendroff Step";
	std::cout << yellow << __func__ << reset;
	nbrOfGrids = 10;
	allocDeviceMemory();
	initDeviceMemory<<<1,16>>>(nbrOfGrids, d_u1, d_u2, d_u3, d_vol, d_h, d_length, d_gama, d_cfl, d_nu, d_tau, d_cMax, d_t);
	laxWendroffStep<<<1,16>>>(nbrOfGrids, d_u1, d_u2, d_u3, d_u1Temp, d_u2Temp, d_u3Temp, 
		d_f1, d_f2, d_f3, d_tau, d_h, d_gama);
	allocHostMemory();
	copyDeviceToHost(nbrOfGrids);
	freeDeviceMemory();
	if((abs(u1[4] - 0.739642857142857) < eps) && (abs(u2[4] - 0.21554331167307) < eps)
		&& (abs(u3[4] - 1.62828130612245) < eps) && (abs(u1[5] - 0.385357142857143) < eps)
		&& (abs(u2[5] - 0.46903163465702) < eps) && (abs(u3[5] - 1.1217186938775515) < eps))
		std::cout << pass << test << reset << std::endl;
	else
		std::cout << fail << test << reset << std::endl;
	freeHostMemory();
}

void ShockTube::DeviceTest04() {
	const std::string test = "Roe Step";
	std::cout << yellow << __func__ << reset;
	nbrOfGrids = 10;
	allocDeviceMemory();
	initDeviceMemory<<<1,16>>>(nbrOfGrids, d_u1, d_u2, d_u3, d_vol, d_h, d_length, d_gama, d_cfl, d_nu, d_tau, d_cMax, d_t);
	RoeStep<<<1,16>>>(nbrOfGrids, d_u1, d_u2, d_u3, d_vol, d_f1, d_f2, d_f3, d_tau, d_h, d_gama,
	w1,w2,w3,w4, fc1,fc2,fc3, fr1,fr2,fr3, fl1,fl2,fl3, fludif1,fludif2,fludif3,
	rsumr, utilde, htilde, uvdif, absvt, ssc, vsc,
	eiglam1,eiglam2,eiglam3, sgn1,sgn2,sgn3, isb1,isb2,isb3, a1,a2,a3, ac11,ac12,ac13, ac21,ac22,ac23);
	allocHostMemory();
	copyDeviceToHost(nbrOfGrids);
	freeDeviceMemory();
	if((abs(u1[4] - 0.702848465455315) < eps) && (abs(u2[4] - 0.342287473165049) < eps)
		&& (abs(u3[4] - 1.5143016216857514) < eps) && (abs(u1[5] - 0.422151534544684) < eps)
		&& (abs(u2[5] - 0.342287473165049) < eps) && (abs(u3[5] - 1.235698378314249) < eps))
		std::cout << pass << test << reset << std::endl;
	else
		std::cout << fail << test << reset << std::endl;
	freeHostMemory();
}

void ShockTube::LaxDevice() {
	std::cout << yellow << __func__ << reset;
	nbrOfGrids = 101;
	allocDeviceMemory();
	initDeviceMemory<<<1,16>>>(nbrOfGrids, d_u1, d_u2, d_u3, d_vol, d_h, d_length, d_gama, d_cfl, d_nu, d_tau, d_cMax, d_t);
	allocHostMemory();
	double tMax = 0.2; t = 0;

	// decrease tau to not overshoot tMax 
	cudaErrorCheck(hipMemcpy(&tau, d_tau, sizeof(double), hipMemcpyDeviceToHost));
	if (tau - tMax > eps) 
		tau = tMax;
	cudaErrorCheck(hipMemcpy(d_tau, &tau, sizeof(double), hipMemcpyHostToDevice));
	int step = 1;
	for(bool tMaxReached = false; tMaxReached==false; step++)
	{
		boundaryCondition<<<1,16>>>(nbrOfGrids, d_u1, d_u2, d_u3);
		updateTau<<<1,1>>>(nbrOfGrids, d_u1, d_u2, d_u3, d_gama, d_cMax, d_h, d_cfl, d_tau); 

		// decrease tau to not overshoot tMax
		cudaErrorCheck(hipMemcpy(&tau, d_tau, sizeof(double), hipMemcpyDeviceToHost));
		if (t + tau - tMax > -eps)
		{ 
			tau = tMax - t;
			tMaxReached = true;
		} 
		cudaErrorCheck(hipMemcpy(d_tau, &tau, sizeof(double), hipMemcpyHostToDevice));

		laxWendroffStep<<<1,16>>>(nbrOfGrids, d_u1, d_u2, d_u3, d_u1Temp, d_u2Temp, d_u3Temp, 
			d_f1, d_f2, d_f3, d_tau, d_h, d_gama);
		t += tau;
	}
	copyDeviceToHost(nbrOfGrids);
	freeDeviceMemory();
	std::ofstream myfile;
	myfile.open("LaxDevice.dat");
	myfile << "variables = x, rho, u, p, mo, e, et, T, c, M, h" << std::endl;
	for (int i = 0; i < nbrOfGrids; i++) {
		double rho = u1[i];
		double u = u2[i] / rho;
		double p = (u3[i] - rho * u * u / 2) * (gama - 1);
		double m = u2[i]; // Momentum I think(?)
		double e = u3[i];
		//double e = p / (gama - 1) / rho; // is this line equivalent to the previous?
		double E = p / (gama - 1.) + 0.5 * rho * u * u;
		double T = p / rho;
		double c = sqrt(gama * p / rho);
		double M = u / c;
		double h = e + p / rho;
		double x = double(i) / double(nbrOfGrids);
		myfile << x << " " << rho << " " << u << " " << p << " " << m << " " << e << " " << E 
			<< " " << T << " " << c << " " << M << " " << h << "\n";
	}
	myfile.close();
	std::cout << blue << "solution: LaxDevice.dat" << reset << std::endl;
	freeHostMemory();
}
