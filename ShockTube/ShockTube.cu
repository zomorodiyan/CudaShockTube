#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <assert.h>
#include <iostream>
#include "ShockTube.cuh"


#define fail "\033[1;31m"
#define reset "\033[0m"
#define cudaErrorCheck(call)                                \
{                                                           \
  hipError_t cucheck_err = (call);                         \
  if(cucheck_err != hipSuccess) {                          \
    const char *err_str = hipGetErrorString(cucheck_err);  \
    printf("\n%s (%d): %s%s%s\n", __FILE__, __LINE__, fail, err_str, reset);\
    assert(0);                                              \
  }                                                         \
}
// Wrap device CUDA calls with cucheck_err as in the following example.
// cudaErrorCheck(hipGetLastError());


// Allocate space for device copies of the variables
void ShockTube::allocDeviceMemory() {
	int size = nbrOfGrids * sizeof(double);
	cudaErrorCheck(hipMalloc((void **)&d_u1, size));
	cudaErrorCheck(hipMalloc((void **)&d_u2, size));
	cudaErrorCheck(hipMalloc((void **)&d_u3, size));
	cudaErrorCheck(hipMalloc((void **)&d_u1Temp, size));
	cudaErrorCheck(hipMalloc((void **)&d_u2Temp, size));
	cudaErrorCheck(hipMalloc((void **)&d_u3Temp, size));
	cudaErrorCheck(hipMalloc((void **)&d_f1, size));
	cudaErrorCheck(hipMalloc((void **)&d_f2, size));
	cudaErrorCheck(hipMalloc((void **)&d_f3, size));
	cudaErrorCheck(hipMalloc((void **)&d_vol, size));
	cudaErrorCheck(hipMalloc((void **)&d_h, sizeof(double)));
	cudaErrorCheck(hipMalloc((void **)&d_length, sizeof(double)));
	cudaErrorCheck(hipMalloc((void **)&d_gama, sizeof(double)));
	cudaErrorCheck(hipMalloc((void **)&d_cfl, sizeof(double)));
	cudaErrorCheck(hipMalloc((void **)&d_nu, sizeof(double)));
	cudaErrorCheck(hipMalloc((void **)&d_tau, sizeof(double)));
	cudaErrorCheck(hipMalloc((void **)&d_cMax, sizeof(double)));
	cudaErrorCheck(hipMalloc((void **)&d_t, sizeof(double)));
}

// Free allocated space for device copies of the variables
void ShockTube::freeDeviceMemory() {
	cudaErrorCheck(hipFree(d_u1));
	cudaErrorCheck(hipFree(d_u2));
	cudaErrorCheck(hipFree(d_u3));
	cudaErrorCheck(hipFree(d_u1Temp));
	cudaErrorCheck(hipFree(d_u2Temp));
	cudaErrorCheck(hipFree(d_u3Temp));
	cudaErrorCheck(hipFree(d_f1));
	cudaErrorCheck(hipFree(d_f2));
	cudaErrorCheck(hipFree(d_f3));
	cudaErrorCheck(hipFree(d_vol));
	cudaErrorCheck(hipFree(d_h));
	cudaErrorCheck(hipFree(d_length));
	cudaErrorCheck(hipFree(d_gama));
	cudaErrorCheck(hipFree(d_cfl));
	cudaErrorCheck(hipFree(d_nu));
	cudaErrorCheck(hipFree(d_tau));
	cudaErrorCheck(hipFree(d_cMax));
}

// calculate and update value of d_cMax
__device__ void updateCMax(const int nbrOfGrids, const double *d_u1,
	const double *d_u2, const double *d_u3, const double *d_gama,
	double *d_cMax) {
	*d_cMax = 0;
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	double ro, p, u;
	__shared__ double c;
	for (int i = index; i < nbrOfGrids; i += stride){
		if (d_u1[i] == 0)
			continue;
		ro = d_u1[i];
		u = d_u2[i] / ro;
		p = (d_u3[i] - ro * u * u / 2) * (*d_gama - 1);
		c = sqrt(*d_gama * abs(p) / ro);
		if (*d_cMax < c + abs(u))
			*d_cMax = c + abs(u);
	}
}

// Assign Sod's shock tube problem initial conditions to device memory
__global__ void initDeviceMemory(const int nbrOfGrids, double *d_u1,
	double *d_u2, double *d_u3, double *d_vol, double *d_h,
	double *d_length, double *d_gama, double *d_cfl, double *d_nu,
	double *d_tau, double *d_cMax, double *d_t) {
	*d_t = 0;							// time
	*d_length = 1;					// length of shock tube
	*d_gama = 1.4;						// ratio of specific heats
	*d_cfl = 0.9;						// Courant-Friedrichs-Lewy number
	*d_nu = 0.0;							// artificial viscosity coefficient
	*d_h = *d_length / (nbrOfGrids - 1);  // space grid size
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x;
	for(int i = index; i < nbrOfGrids; i+= stride){
		double e, ro, p, u = 0;
		if (index < nbrOfGrids){
			if (index >= int(nbrOfGrids / 2)) { ro = 0.125, p = 0.1; }
			else { ro = 1, p = 1; }
			e = p / (*d_gama - 1) + ro * u * u / 2;
			d_u1[i] = ro;
			d_u2[i] = ro * u;
			d_u3[i] = e;
			d_vol[i] = 1;
		}
	}
	updateCMax(nbrOfGrids, d_u1, d_u2, d_u3, d_gama, d_cMax); 
	*d_tau = (*d_cfl) * (*d_h) / (*d_cMax);    // time grid size
}

// copy device data members to host data members
void ShockTube::copyDeviceToHost(const int nbrOfGrids) {
	int size = nbrOfGrids * sizeof(double);
	cudaErrorCheck(hipMemcpy(u1, d_u1, size, hipMemcpyDeviceToHost));
	cudaErrorCheck(hipMemcpy(u2, d_u2, size, hipMemcpyDeviceToHost));
	cudaErrorCheck(hipMemcpy(u3, d_u3, size, hipMemcpyDeviceToHost));
	cudaErrorCheck(hipMemcpy(vol, d_vol, size, hipMemcpyDeviceToHost));
	cudaErrorCheck(hipMemcpy(&h, d_h, sizeof(double), hipMemcpyDeviceToHost));
	cudaErrorCheck(hipMemcpy(&length, d_length, sizeof(double), hipMemcpyDeviceToHost));
	cudaErrorCheck(hipMemcpy(&gama, d_gama, sizeof(double), hipMemcpyDeviceToHost));
	cudaErrorCheck(hipMemcpy(&cfl, d_cfl, sizeof(double), hipMemcpyDeviceToHost));
	cudaErrorCheck(hipMemcpy(&cMax, d_cMax, sizeof(double), hipMemcpyDeviceToHost));
	cudaErrorCheck(hipMemcpy(&nu, d_nu, sizeof(double), hipMemcpyDeviceToHost));
	cudaErrorCheck(hipMemcpy(&tau, d_tau, sizeof(double), hipMemcpyDeviceToHost));
}

// copy flux from device to host (for debegging purpose)
void ShockTube::copyFluxFromDeviceToHost(const int nbrOfGrids) {
	int size = nbrOfGrids * sizeof(double);
	cudaErrorCheck(hipMemcpy(f1, d_f1, size, hipMemcpyDeviceToHost));
	cudaErrorCheck(hipMemcpy(f2, d_f2, size, hipMemcpyDeviceToHost));
	cudaErrorCheck(hipMemcpy(f3, d_f3, size, hipMemcpyDeviceToHost));
}


// copy host data members to device data members
void ShockTube::copyHostToDevice(const int nbrOfGrids) {
	int size = nbrOfGrids * sizeof(double);
	cudaErrorCheck(hipMemcpy(d_u1, u1, size, hipMemcpyHostToDevice));
	cudaErrorCheck(hipMemcpy(d_u2, u2, size, hipMemcpyHostToDevice));
	cudaErrorCheck(hipMemcpy(d_u3, u3, size, hipMemcpyHostToDevice));
	/*/
	cudaErrorCheck(hipMemcpy(d_f1, f1, size, hipMemcpyHostToDevice));
	cudaErrorCheck(hipMemcpy(d_f2, f2, size, hipMemcpyHostToDevice));
	cudaErrorCheck(hipMemcpy(d_f3, f3, size, hipMemcpyHostToDevice));
	/**/
	cudaErrorCheck(hipMemcpy(d_vol, vol, size, hipMemcpyHostToDevice));
	cudaErrorCheck(hipMemcpy(d_h, &h, sizeof(double), hipMemcpyHostToDevice));
	cudaErrorCheck(hipMemcpy(d_length, &length, sizeof(double), hipMemcpyHostToDevice));
	cudaErrorCheck(hipMemcpy(d_gama, &gama, sizeof(double), hipMemcpyHostToDevice));
	cudaErrorCheck(hipMemcpy(d_cfl, &cfl, sizeof(double), hipMemcpyHostToDevice));
	cudaErrorCheck(hipMemcpy(d_nu, &nu, sizeof(double), hipMemcpyHostToDevice));
	cudaErrorCheck(hipMemcpy(d_tau, &tau, sizeof(double), hipMemcpyHostToDevice));
}


__global__ void boundaryCondition(const int nbrOfGrids,
	double *d_u1, double *d_u2, double *d_u3) {
	d_u1[0] = d_u1[1];
	d_u2[0] = -d_u2[1];
	d_u3[0] = d_u3[1];
	d_u1[nbrOfGrids - 1] = d_u1[nbrOfGrids - 2];
	d_u2[nbrOfGrids - 1] = -d_u2[nbrOfGrids - 2];
	d_u3[nbrOfGrids - 1] = d_u3[nbrOfGrids - 2];
}

__global__ void updateTau(const int nbrOfGrids, const double *d_u1,
	const double *d_u2, const double *d_u3, const double *d_gama,
	double *d_cMax, const double *d_h, const double *d_cfl, double *d_tau) {
	updateCMax(nbrOfGrids, d_u1, d_u2, d_u3, d_gama, d_cMax);
	*d_tau = *d_cfl * *d_h / *d_cMax;
}

// used in laxWendroffStep
__device__ void d_boundaryCondition(const int nbrOfGrids,
	double *d_u1, double *d_u2, double *d_u3) {
	d_u1[0] = d_u1[1];
	d_u2[0] = -d_u2[1];
	d_u3[0] = d_u3[1];
	d_u1[nbrOfGrids - 1] = d_u1[nbrOfGrids - 2];
	d_u2[nbrOfGrids - 1] = -d_u2[nbrOfGrids - 2];
	d_u3[nbrOfGrids - 1] = d_u3[nbrOfGrids - 2];
}

// used in laxWendroffStep
__device__ void updateFlux(const int nbrOfGrids, const double *d_u1, const double *d_u2,
	const double *d_u3, double *d_f1, double *d_f2, double *d_f3, const double *d_gama) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x;
	double rho, m, e, p;
	for (int i = index; i < nbrOfGrids; i += stride) {
		rho = d_u1[i];
		m = d_u2[i];
		e = d_u3[i];
		p = (*d_gama - 1) * (e - m * m / rho / 2);
		d_f1[i] = m;
		d_f2[i] = m * m / rho + p;
		d_f3[i] = m / rho * (e + p);
	}
}

// used in laxWendroffStep
__device__ void halfStep(const int nbrOfGrids, const double *d_u1, const double *d_u2,
	const double *d_u3, double *d_u1Temp, double *d_u2Temp, double *d_u3Temp,
	const double *d_f1, const double *d_f2, const double *d_f3, const double *d_tau, const double *d_h) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x;
	for (int i = index; i < nbrOfGrids; i += stride) {
		if ((i > 0) && (i < nbrOfGrids - 1)) {
			d_u1Temp[i] = (d_u1[i + 1] + d_u1[i]) / 2 - *d_tau / 2 / *d_h * (d_f1[i + 1] - d_f1[i]);
			d_u2Temp[i] = (d_u2[i + 1] + d_u2[i]) / 2 - *d_tau / 2 / *d_h * (d_f2[i + 1] - d_f2[i]);
			d_u3Temp[i] = (d_u3[i + 1] + d_u3[i]) / 2 - *d_tau / 2 / *d_h * (d_f3[i + 1] - d_f3[i]);
		}
	}
}

// used in laxWendroffStep
__device__ void step(const int nbrOfGrids, const double *d_u1, const double *d_u2,
	const double *d_u3, double *d_u1Temp, double *d_u2Temp, double *d_u3Temp,
	const double *d_f1, const double *d_f2, const double *d_f3, const double *d_tau, const double *d_h) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x;
	for (int i = index; i < nbrOfGrids; i += stride) {
		if ((i > 0) && (i < nbrOfGrids - 1)) {
			d_u1Temp[i] = d_u1[i] - *d_tau / *d_h * (d_f1[i] - d_f1[i - 1]);
			d_u2Temp[i] = d_u2[i] - *d_tau / *d_h * (d_f2[i] - d_f2[i - 1]);
			d_u3Temp[i] = d_u3[i] - *d_tau / *d_h * (d_f3[i] - d_f3[i - 1]);
		}
	}
}

// used in laxWendroffStep
__device__ void updateU(const int nbrOfGrids, double *d_u1, double *d_u2,
	double *d_u3, const double *d_u1Temp, const double *d_u2Temp, const double *d_u3Temp) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x;
	for (int i = index; i < nbrOfGrids; i += stride) {
		if ((i > 0) && (i < nbrOfGrids - 1)) {
			d_u1[i] = d_u1Temp[i];
			d_u2[i] = d_u2Temp[i];
			d_u3[i] = d_u3Temp[i];
		}
	}
}

__global__	void laxWendroffStep(const int nbrOfGrids, double *d_u1, double *d_u2,
	double *d_u3, double *d_u1Temp, double *d_u2Temp, double *d_u3Temp,
	double *d_f1, double *d_f2, double *d_f3, const double *d_tau, const double *d_h, const double *d_gama) {
	updateFlux(nbrOfGrids, d_u1, d_u2, d_u3, d_f1, d_f2, d_f3, d_gama);
	halfStep(nbrOfGrids, d_u1, d_u2, d_u3, d_u1Temp, d_u2Temp, d_u3Temp, d_f1, d_f2, d_f3, d_tau, d_h);
	d_boundaryCondition(nbrOfGrids, d_u1Temp, d_u2Temp, d_u3Temp);
	updateFlux(nbrOfGrids, d_u1Temp, d_u2Temp, d_u3Temp, d_f1, d_f2, d_f3, d_gama);
	step(nbrOfGrids, d_u1, d_u2, d_u3, d_u1Temp, d_u2Temp, d_u3Temp, d_f1, d_f2, d_f3, d_tau, d_h);
	updateU(nbrOfGrids, d_u1, d_u2, d_u3, d_u1Temp, d_u2Temp, d_u3Temp);
}


__global__	void RoeStep(const int nbrOfGrids, double *d_u1, double *d_u2,
	double *d_u3, double *d_u1Temp, double *d_u2Temp, double *d_u3Temp,
	double *d_f1, double *d_f2, double *d_f3, const double *d_tau, const double *d_h, const double *d_gama) {
	;
}
